#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#define MSG(a) std::cout << #a << " " << a << std::endl;

int inv(const float *A, float* B, int n)
{
    hipsolverStatus_t status, status0, status1, status2;
    hipsolverHandle_t handle;
    status = hipsolverDnCreate(&handle);
    
    mkide(B, n);
    float *A_;
    hipMalloc(&A_, sizeof(float) * n * n);
    hipMemcpy(A_, A, sizeof(float) * n * n, hipMemcpyDeviceToDevice);
    int worksize;
    float *workspace;
    int *devInfo;
    int *devIpiv;
    hipMalloc(&devInfo, sizeof(int));
    hipMalloc(&devIpiv, sizeof(int)*n);

    status0 = hipsolverDnSgetrf_bufferSize(handle,
        n, n,
        A_,
        n,
        &worksize);
    hipMalloc(&workspace, sizeof(float)*worksize);
    
    status1 = hipsolverDnSgetrf(handle,
        n, n,
        A_,
        n,
        workspace,
        devIpiv,
        devInfo);

    status2 = hipsolverDnSgetrs(handle,
        HIPBLAS_OP_N,
        n,
        n,
        A_,
        n,
        devIpiv,
        B,
        n,
        devInfo);
        hipsolverDnDestroy(handle);
    return status+status0+status1+status2;
}
